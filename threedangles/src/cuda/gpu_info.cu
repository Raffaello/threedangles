#include <cuda/gpu_info.cuh>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/version.h>

void display_gpu_info()
{
    using std::cout;
    using std::endl;
    using std::cerr;
    
    cout << "gpu info" << endl;
    
    int devCount;
    if (hipGetDeviceCount(&devCount) != hipSuccess) {
        cerr << hipGetLastError() << endl;
        return;
    }
    cout << "CUDA :" << CUDART_VERSION << " - THRUST: " << THRUST_VERSION << endl;
    cout << "CUDA device count:" << devCount << endl;
    for (int i = 0; i < devCount; i++)
    {
        hipDeviceProp_t devProp;
        if (hipGetDeviceProperties(&devProp, i) != hipSuccess) {
            cerr << hipGetLastError() << endl;
            continue;
        }

        cout
            << "Device:     " << i << endl
            << "Name:       " << devProp.name << " - " << devProp.major << "." << devProp.minor << endl
            << "Global Mem: " << devProp.totalGlobalMem << endl
            << "Shared Mem: " << devProp.sharedMemPerBlock << endl
            ;
    }
}

