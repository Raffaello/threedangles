#include <cuda/GPUInfo.cuh>
#include <cassert>

namespace cuda
{
    GPUInfo::GPUInfo() : cudart_version(CUDART_VERSION), thrust_version(THRUST_VERSION)
    {
        int deviceCount;
        if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
            _errors.push_back(hipGetErrorString(hipGetLastError()));
            return;
        }

        for (int i = 0; i < deviceCount; i++)
        {
            hipDeviceProp_t devProp;
            if (hipGetDeviceProperties(&devProp, i) != hipSuccess) {
                _errors.push_back(hipGetErrorString(hipGetLastError()));
                continue;
            }

            _deviceProperties.push_back(devProp);
        }

        assert(deviceCount == _deviceProperties.size());
    }

    int GPUInfo::getErrorsCount() const noexcept
    {
        return _errors.size();
    }

    thrust::host_vector<std::string> GPUInfo::getErrors() const noexcept
    {
        return _errors;
    }

    int GPUInfo::getDeviceCount() const noexcept
    {
        return _deviceProperties.size();
    }

    thrust::host_vector<hipDeviceProp_t> GPUInfo::getDeviceProperties() const noexcept
    {
        return _deviceProperties;
    }
}
